
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
   for(long j = 0; j < 20; j++)
	{
		int i = index; 
		x[i] = y[i] + 2;
		y[i] = x[i] * 7;				
	}	
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
	int blockSize = 256;
	int numBlocks = 100;
	add<<<numBlocks, blockSize>>>(N, x, y);
	
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
	for(int i = 0; i < 10; i++)
		std::cout << x[i];
	
	std::cout << "Hello!";

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}