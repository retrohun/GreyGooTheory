
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define cuda __device__  
#define kernel __global__  
#define common __host__  

template<class T>
common T* cu_new_arr(int size)
{
    T* result; 
	hipMallocManaged((void**)&result, size * sizeof(T));
	//cudaMemset(result, 0, size * sizeof(T));
	return result; 
}

kernel
void add(int n, int *x, int *y)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//int xi = x[i];
	//int yi = y[i];
	int i = idx; 
	
	for(long j = 0; j < 50000000; j++)
	{
		
		 i += 17;
		 
		 if(i > 1000000)
			 i /= 2; 
		 
		 //y[i] = x[i] + 17; 
	}	
	
	x[idx] = i;
	//y[i] = yi; 
}

int main(void)
{
	int N = 1<<20;
	int *x, *y;
	x = cu_new_arr<int>(N);
	y = cu_new_arr<int>(N);

	for (int i = 0; i < N; i++) {
		x[i] = 1;
		y[i] = 2;
	}

	int blockSize = 32;
	int numBlocks = 18;
	add<<<numBlocks, blockSize>>>(N, x, y);

	hipDeviceSynchronize();
	std::cout << numBlocks << "," << blockSize <<"\n\n\n";

	hipFree(x);
	hipFree(y);

	return 0;
}

//50000 * 32 * 18 rea/write + calculate = 1 sec. 
//10 operations. 
//