#include <iostream>
#include <math.h>
#include "utils.cuh"
#include "snake.cuh"

kernel
void add(int n, LifeState *states)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	states[idx].data[0] = states[idx].W + states[idx].H; 
}

int main(void)
{
	int blockSize = 32;
	int numBlocks = 18;
	
	int N = blockSize * numBlocks;
	LifeState* states;
	states = cu_new_arr<LifeState>(N);
	
	for(int i = 0; i < N; i++)
		states[i].Init(10,10);
	
	add<<<numBlocks, blockSize>>>(N, states);

	hipDeviceSynchronize();
	
	std::cout << states[0].data[0];
	
	for(int i = 0; i < N; i++)
	{	
		std::cout << states[i].data[0];
		states[i].~LifeState();
	}
	hipFree(states);
	
	return 0;
}